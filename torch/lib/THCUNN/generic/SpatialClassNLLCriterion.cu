
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialClassNLLCriterion.cu"
#else

void THNN_(SpatialClassNLLCriterion_shapeCheck)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *weights)
{
  THArgCheck(THCIndexTensor_(nDimension)(state, target) == 3, 1,
             "only batches of spatial targets supported (3D tensors)" \
             " but got targets of dimension: %d",
             THCIndexTensor_(nDimension)(state, target));
  THArgCheck(THCTensor_(nDimension)(state, input) == 4, 2,
             "only batches of spatial inputs supported (4D tensors), "      \
             "but got input of dimension: %d", THCTensor_(nDimension)(state, input));
  if (THCTensor_(size)(state, input, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, input, 2) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, input, 3) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff input_size = THCTensor_(sizeDesc)(state, input);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("input and target batch or spatial sizes don't match: target %s, input %s",
            target_size.str, input_size.str);
  }

  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }
}

void THNN_(SpatialClassNLLCriterion_gradOutput_no_reduce_shapeCheck)(
           THCState *state,
           THCTensor *gradOutput,
           THCIndexTensor *target)
{
  THArgCheck(THCTensor_(nDimension)(state, gradOutput) == 3, 2,
             "Expected dimension 3 but got gradOutput of dimension: %d",
             THCTensor_(nDimension)(state, gradOutput));
  if (THCTensor_(size)(state, gradOutput, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, gradOutput, 1) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, gradOutput, 2) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff gradOutput_size = THCTensor_(sizeDesc)(state, gradOutput);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("gradOutput sizes don't match target sizes: target %s, gradOutput %s",
            target_size.str, gradOutput_size.str);
  }
}

void THNN_(SpatialClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight,
           int64_t ignore_index,
           bool reduce)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THCTensor_(resize1d)(state, output, 1);

  if (weights)
    THCUNN_assertSameGPU(state, 5, input, target, weights, output, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, output, total_weight);

  if (!reduce) {
    int64_t batch_size = THCTensor_(size)(state, input, 0);
    int64_t H = THCTensor_(size)(state, input, 2);
    int64_t W = THCTensor_(size)(state, input, 3);
    
    THCTensor_(resize3d)(state, output, batch_size, H, W);

    bool not_contiguous_weights = 
        weights && !THCTensor_(isContiguous)(state, weights);
    if (not_contiguous_weights) {
      weights = THCTensor_(newContiguous)(state, weights);
    }

    int64_t count = batch_size * H * W;
    SpatialClassNLLCriterion_updateOutput_no_reduce_kernel<real>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
        count, 
        toDeviceTensor<real, 4>(state, input), 
        toDeviceTensor<THCIndex_t, 3>(state, target), 
        toDeviceTensor<real, 3>(state, output), 
        weights ? THCTensor_(data)(state, weights) : NULL, 
        ignore_index);

    if (not_contiguous_weights) {
      THCTensor_(free)(state, weights);
    }
    return;
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *input_data = THCTensor_(data)(state, input);
  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *output_data = THCTensor_(data)(state, output);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  THCTensor_(fill)(state, output, ScalarConvert<int, real>::to(0));
  THCTensor_(fill)(state, total_weight, ScalarConvert<int, real>::to(0));

  cunn_SpatialClassNLLCriterion_updateOutput_kernel<real, accreal>
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      output_data,
      total_weight_data,
      input_data,
      target_data,
      weights_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) * THCTensor_(size)(state, input, 3),
      blocks_per_sample,
      ignore_index
  );
  THCudaCheck(cudaGetLastError());
  if (sizeAverage) {
    cunn_SpatialClassNLLCriterion_sizeAverage_kernel<<<1, 1, 0, THCState_getCurrentStream(state)>>>(
      output_data, total_weight_data
    );
    THCudaCheck(cudaGetLastError());
  }

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(SpatialClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight,
           int64_t ignore_index,
           bool reduce)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4,
             "gradInput must be contiguous");

  if (weights)
    THCUNN_assertSameGPU(state, 5, weights, input, target, gradInput, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, gradInput, total_weight);

  if (!reduce) {
    THNN_(SpatialClassNLLCriterion_gradOutput_no_reduce_shapeCheck)(
        state, 
        gradOutput, 
        target);

    int64_t batch_size = THCTensor_(size)(state, input, 0);
    int64_t H = THCTensor_(size)(state, input, 2);
    int64_t W = THCTensor_(size)(state, input, 3);

    bool not_contiguous_weights = 
        weights && !THCTensor_(isContiguous)(state, weights);
    if (not_contiguous_weights) {
      weights = THCTensor_(newContiguous)(state, weights);
    }

    int64_t count = batch_size * H * W;
    SpatialClassNLLCriterion_updateGradInput_no_reduce_kernel<real>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
        count, 
        toDeviceTensor<THCIndex_t, 3>(state, target), 
        toDeviceTensor<real, 3>(state, gradOutput), 
        toDeviceTensor<real, 4>(state, gradInput), 
        weights ? THCTensor_(data)(state, weights) : NULL, 
        ignore_index);

    if (not_contiguous_weights) {
      THCTensor_(free)(state, weights);
    }
    return;
  }  
  
  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t *target_data = THCIndexTensor_(data)(state, target);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  cunn_SpatialClassNLLCriterion_updateGradInput_kernel
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      gradInput_data,
      THCTensor_(get1d)(state, gradOutput, 0),
      target_data,
      weights_data,
      total_weight_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) *THCTensor_(size)(state, input, 3),
      blocks_per_sample,
      ignore_index
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

#undef TO_DEVICE

#endif
